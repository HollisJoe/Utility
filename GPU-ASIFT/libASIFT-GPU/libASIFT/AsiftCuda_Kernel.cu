#include "hip/hip_runtime.h"
#include "AsiftCuda_Kernel.h"

#define MAX_C(i,j) ( (i)<(j) ? (j):(i) )
#define MIN_C(i,j) ( (i)<(j) ? (i):(j) )
#define ABS_C(x)   (((x) > 0) ? (x) : (-(x)))

texture<float> texOriImg;

__device__ __constant__ float kernel_c[MAX_KERNEL_SIZE_CU];

using namespace std;

void unbindTexture_texOriImg() {
	hipUnbindTexture(texOriImg);
}

void bindTexture_texOriImg(float* img_dev, const float const* img_host, int width, int height) {
	hipBindTexture(NULL, texOriImg, img_dev, width*height*sizeof(float));
	hipMemcpy(img_dev, img_host, width*height*sizeof(float), hipMemcpyHostToDevice);
}

__global__ void frot_cu(float* data_out,float s,float c,int xi,int xa,int yi,int ya,int xo,int yo,int xn) {
	__shared__ int xmin, xmax, ymin, ymax;
	__shared__ int nx, ny, sx;
	__shared__ float sa, ca;
	if(threadIdx.x == 0 && threadIdx.y == 0) {
	xmin = xi;	xmax = xa;	ymin = yi;	ymax = ya;
	nx	 = xo;  ny   = yo;  sx   = xn;  sa   = s;	ca   = c;
	}
	__syncthreads();

	int x_bias = xmin + threadIdx.x + blockIdx.x * blockDim.x;
	int y_bias = ymin + threadIdx.y + blockIdx.y * blockDim.y;
	if(x_bias<=xmax && y_bias<=ymax) {
		float xp = ca * (float)x_bias - sa * (float)y_bias;
		float yp = sa * (float)x_bias + ca * (float)y_bias;
		int x1 = (int)floor(xp);
		int y1 = (int)floor(yp);
		float ux = xp - (float)x1;
		float uy = yp - (float)y1;
		int adr = y1 * nx + x1;
		int tx1 = (x1>=0 && x1<nx);
		int tx2 = (x1+1>=0 && x1+1<nx);
		int ty1 = (y1>=0 && y1<ny);
		int ty2 = (y1+1>=0 && y1+1<ny);

		float a11 = (tx1 && ty1? tex1Dfetch(texOriImg, adr) : 128);
		float a12 = (tx1 && ty2? tex1Dfetch(texOriImg, adr+nx) : 128);
		float a21 = (tx2 && ty1? tex1Dfetch(texOriImg, adr+1) : 128);
		float a22 = (tx2 && ty2? tex1Dfetch(texOriImg, adr+nx+1) : 128);

		data_out[(y_bias-ymin)*sx+x_bias-xmin] = 
			(1.0-uy)*((1.0-ux)*a11+ux*a21)+uy*((1.0-ux)*a12+ux*a22);
	}
}

template<int COLUMNS_BLOCKDIM_X, int COLUMNS_BLOCKDIM_Y>
__global__ void convolutionColumnsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{
	__shared__ int image_size;
    __shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];
	int tmp;
	if(threadIdx.x == 0 && threadIdx.y == 0) image_size = imageW * imageH;
	__syncthreads();

    //Offset to the upper halo edge
    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
	if(baseX >= imageW) goto out_of_bound;
    const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
	const int upper_initial = baseY * pitch + baseX;
	d_Src += upper_initial;
	d_Dst += upper_initial;

    //Main data
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
		tmp = i * COLUMNS_BLOCKDIM_Y * pitch;
		s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (upper_initial + tmp < image_size)? d_Src[tmp] : 0;
    }

    //Upper halo
#pragma unroll

    for (int i = 0; i < COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
    }

    //Lower halo
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
    {
		tmp = i * COLUMNS_BLOCKDIM_Y * pitch;
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (upper_initial + tmp < image_size)? d_Src[tmp] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
		tmp = i * COLUMNS_BLOCKDIM_Y * pitch;
		if(upper_initial + tmp >= image_size) break;
        float sum = 0;
#pragma unroll

        for (int j = -COLUMNS_BLOCKDIM_Y; j <= COLUMNS_BLOCKDIM_Y; j++)
        {
            sum += kernel_c[COLUMNS_BLOCKDIM_Y - j] * s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
        }

        d_Dst[tmp] = sum;
    }
out_of_bound:
}

__device__ void compensate_affine_coor_CU(float* x0, float* y0, int w1, int h1, float t1, float t2, float Rtheta) {
	float x_ori, y_ori;
	float x1 = *x0, y1 = *y0;
	Rtheta = Rtheta * PI_CU / 180;
	if(Rtheta <= 1.57079632679489661923) {
		x_ori = 0;
		y_ori = w1 * sin(Rtheta) / t1;
	} else {
		x_ori = -w1 * cos(Rtheta) / t2;
		y_ori = ( w1 * sin(Rtheta) + h1 * sin(Rtheta-1.57079632679489661923) ) / t1;
	}
	float sin_Rtheta = sin(Rtheta);
	float cos_Rtheta = cos(Rtheta);
	x1	= (x1 - x_ori) * t2;
	y1	= (y1 - y_ori) * t1;
	*x0 = cos_Rtheta * x1 - sin_Rtheta * y1;
	*y0 = sin_Rtheta * x1 + cos_Rtheta * y1;
}

__global__ void filterKernel(
	float4* keys,
	int* index, int* filtered_num,
	int feature_numG, int widthG, int heightG,
	float sin_thetaG, float cos_thetaG, float thetaG, float t2G
	) 
{
	__shared__ int   blockCounter;
	__shared__ int	 feature_num, width, height;
	__shared__ float sin_theta1, cos_theta1, theta, t2;
	if(threadIdx.x == 0) {
		feature_num = feature_numG; blockCounter = 0;
		width		= widthG;		height		 = heightG;
		sin_theta1	= sin_thetaG;	cos_theta1	 = cos_thetaG;
		theta		= thetaG;		t2			 = t2G;
	}
	__syncthreads();

	int x_id = blockIdx.x * blockDim.x + threadIdx.x;
	if(x_id < feature_num) {
		float x0, y0, x1, y1, x2, y2, x3, y3, x4, y4, d1, d2, d3, d4, BorderTh;
		float4 current_key = keys[x_id];
		x0 = current_key.x;
		y0 = current_key.y;
		if(theta <= 90) {
			x1 = height * sin_theta1;
			y1 = 0;			 
			y2 = width * sin_theta1;
			x3 = width * cos_theta1;
			x4 = 0;
			y4 = height * cos_theta1;
			x2 = x1 + x3;
			y3 = y2 + y4;
			y1 = y3 - y1;
			y2 = y3 - y2;
			y4 = y3 - y4;
			y3 = 0;
			y1 = y1 * t2;
			y2 = y2 * t2;
			y3 = y3 * t2;
			y4 = y4 * t2;
		} else {
			y1 = -height * cos_theta1;
			x2 = height * sin_theta1;
			x3 = 0;
			y3 = width * sin_theta1;				 
			x4 = -width * cos_theta1;
			y4 = 0;
			x1 = x2 + x4;
			y2 = y1 + y3;
			y1 = y2 - y1;
			y3 = y2 - y3;
			y4 = y2 - y4;
			y2 = 0;
			y1 = y1 * t2;
			y2 = y2 * t2;
			y3 = y3 * t2;
			y4 = y4 * t2;
		}
		d1 = ABS_C((x2-x1)*(y1-y0)-(x1-x0)*(y2-y1)) / sqrt((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1));
		d2 = ABS_C((x3-x2)*(y2-y0)-(x2-x0)*(y3-y2)) / sqrt((x3-x2)*(x3-x2)+(y3-y2)*(y3-y2));
		d3 = ABS_C((x4-x3)*(y3-y0)-(x3-x0)*(y4-y3)) / sqrt((x4-x3)*(x4-x3)+(y4-y3)*(y4-y3));
		d4 = ABS_C((x1-x4)*(y4-y0)-(x4-x0)*(y1-y4)) / sqrt((x1-x4)*(x1-x4)+(y1-y4)*(y1-y4));
		BorderTh = 8.4852813 * current_key.z;
		if (!((d1<BorderTh) || (d2<BorderTh) || (d3<BorderTh) || (d4<BorderTh) )) {
			compensate_affine_coor_CU(&x0, &y0, width, height, 1/t2, 1.0, theta);
			keys[x_id].x = x0;
			keys[x_id].y = y0;
			index[x_id]	 = 1;
			atomicAdd(&blockCounter, 1);
		}
		else index[x_id] = 0;
	}
	__syncthreads();
	if(threadIdx.x == 0)
		atomicAdd(filtered_num, blockCounter);
}

void generate_kernel(float*& kernel_h, int& ksize, float sigma) {
	float x, sum = 0.0;
	ksize = (int)(2.0 * GaussTruncate_cu * sigma + 1.0);
	ksize = MAX_C(3, ksize);
	if(ksize%2 == 0) ksize++;
	assert(ksize < MAX_KERNEL_SIZE_CU);
	kernel_h = (float*)malloc(ksize * sizeof(float));
	for(int i=0; i<ksize; i++) {
		x = i - ksize / 2;
		kernel_h[i] = exp(-x * x / (2.0 * sigma * sigma));
		sum += kernel_h[i];
	}
	for(int i=0; i<ksize; i++) kernel_h[i] /= sum;
}

void bound_CU(int x, int y, float ca, float sa, int *xmin, int *xmax, int *ymin, int *ymax)
{   
    int rx,ry;
	
    rx = (int)floor(ca*(float)x+sa*(float)y);
    ry = (int)floor(-sa*(float)x+ca*(float)y);
    if (rx<*xmin) *xmin=rx; if (rx>*xmax) *xmax=rx;
    if (ry<*ymin) *ymin=ry; if (ry>*ymax) *ymax=ry;
}

template<int COLUMNS_BLOCKDIM_X, int COLUMNS_BLOCKDIM_Y>
void convolutionVerticalCuda(float*& rot_d, float*& blurred_d, int width, int height) {
	hipMalloc((void**)&blurred_d, height*width*sizeof(float));
	dim3 blocks((width+COLUMNS_BLOCKDIM_X-1)/COLUMNS_BLOCKDIM_X, (height+COLUMNS_RESULT_STEPS*COLUMNS_BLOCKDIM_Y-1)/(COLUMNS_RESULT_STEPS*COLUMNS_BLOCKDIM_Y));
	dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);
	convolutionColumnsKernel<COLUMNS_BLOCKDIM_X,COLUMNS_BLOCKDIM_Y><<<blocks, threads>>>(blurred_d, rot_d, width, height, width);
	hipFree(rot_d);
}

void frotCuda(float*& output_d, int width, int height, int& widthO, int& heightO, float* theta) {
	int xmin=0, xmax=0, ymin=0, ymax=0;
	float ca = (float)cos((double)(*theta)*PI_CU/180.0);
	float sa = (float)sin((double)(*theta)*PI_CU/180.0);
	bound_CU(width-1,0,ca,sa,&xmin,&xmax,&ymin,&ymax);
	bound_CU(0,height-1,ca,sa,&xmin,&xmax,&ymin,&ymax);
	bound_CU(width-1,height-1,ca,sa,&xmin,&xmax,&ymin,&ymax);
	widthO = xmax - xmin + 1;
	heightO = ymax - ymin + 1;
	hipMalloc((void**)&output_d, heightO*widthO*sizeof(float));

	dim3 blocks((widthO+15)/16, (heightO+15)/16);
	dim3 threads(16, 16);
	frot_cu<<<blocks, threads>>>(output_d, sa, ca, xmin, xmax, ymin, ymax, width, height, widthO);
}

void gaussianVerticalCuda(float*& rot_d, float*& blurred_d, int width, int height, float sigma) {
	int size_kernel;
	float* kernel_h;
	generate_kernel(kernel_h, size_kernel, sigma);
	hipMemcpyToSymbol(HIP_SYMBOL(kernel_c), kernel_h, size_kernel*sizeof(float));
	free(kernel_h);

	switch(size_kernel) {
	case 11:	convolutionVerticalCuda<32,5>(rot_d, blurred_d, width, height); break;
	case 13:	convolutionVerticalCuda<32,6>(rot_d, blurred_d, width, height); break;
	case 19:	convolutionVerticalCuda<16,9>(rot_d, blurred_d, width, height); break;
	case 27:	convolutionVerticalCuda<16,13>(rot_d, blurred_d, width, height); break;
	case 37:	convolutionVerticalCuda<16,18>(rot_d, blurred_d, width, height); break;
	case 53:	convolutionVerticalCuda<8,26>(rot_d, blurred_d, width, height); break;
	default:	break;
	}
}

void warpCuda(float*& blurred_d, float*& warped_d, float t1, float t2, int widthRot, int heightRot, int widthTilt, int heightTilt) {
	NppiSize src_size = {widthRot, heightRot};
	NppiRect src_roi  = {0,0,widthRot,heightRot};
	NppiRect dst_roi  = {0,0,widthTilt,heightTilt};
	int pitchIn  = widthRot * sizeof(float);
	int pitchOut = widthTilt * sizeof(float);
	double coeffs[2][3] = {{t1,0,0},{0,t2,0}};
	hipMalloc((void**)&warped_d, heightTilt*widthTilt*sizeof(float));
	nppiWarpAffine_32f_C1R((Npp32f*)blurred_d, src_size, pitchIn, src_roi, (Npp32f*)warped_d, pitchOut, dst_roi, coeffs, NPPI_INTER_LINEAR);
	hipFree(blurred_d);
}

void filterPoints(
	float4* keys_host,
	int* index_host, int* filtered_num_host,
	int feature_num, int width, int height,
	float sin_theta1, float cos_theta1, float theta, float t2
	)
{
	int*	index_dev, *filtered_num_dev;
	float4* keys_dev;
	hipMalloc((void**)&filtered_num_dev	, sizeof(int));
	hipMalloc((void**)&index_dev			, feature_num * sizeof(int));
	hipMalloc((void**)&keys_dev			, feature_num * sizeof(float4));
	hipMemcpy(keys_dev, keys_host, feature_num * sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(filtered_num_dev, filtered_num_host, sizeof(int), hipMemcpyHostToDevice);
	filterKernel<<<(feature_num+255)/256, 256>>>(keys_dev, index_dev, filtered_num_dev,
		feature_num, width, height, sin_theta1, cos_theta1, theta, t2);
	hipMemcpy(keys_host, keys_dev, feature_num * sizeof(float4), hipMemcpyDeviceToHost);
	hipMemcpy(index_host, index_dev, feature_num * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(filtered_num_host, filtered_num_dev, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(index_dev);
	hipFree(filtered_num_dev);
	hipFree(keys_dev);
}